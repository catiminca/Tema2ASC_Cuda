#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "../include/utils.cuh"
#include <string.h>
#include <stdlib.h>
#include <inttypes.h>

// TODO: Implement function to search for all nonces from 1 through MAX_NONCE (inclusive) using CUDA Threads
__global__ void findNonce(int number_th, BYTE* block_content, size_t current_length, BYTE* block_hash, uint64_t* nonce_final, BYTE* difficulty_5_zeros, int* flag) {
	unsigned int th = threadIdx.x + blockDim.x * blockIdx.x;
	uint64_t start = th * (double)MAX_NONCE / number_th;
	uint64_t end = (th + 1) * MAX_NONCE / number_th;
	if (end > MAX_NONCE) {
		end = MAX_NONCE;
	}
	BYTE block_content_copy[BLOCK_SIZE], block_hash_copy[SHA256_HASH_SIZE];
	d_strcpy((char*) block_content_copy, (const char*) block_content);
	for (uint64_t nonce = start; nonce < end; nonce++) {
		char nonce_string[NONCE_SIZE];
		intToString(nonce, nonce_string);
		d_strcpy((char*) block_content_copy + current_length, nonce_string);
        apply_sha256(block_content_copy, d_strlen((const char*)block_content_copy), block_hash_copy, 1);
		if (compare_hashes(block_hash_copy, difficulty_5_zeros) <= 0) {
				atomicAdd(flag, 1);
				if (*flag == 1) {
					*nonce_final = nonce;
					d_strcpy((char*) block_hash, (const char*) block_hash_copy);
					d_strcpy((char*) block_content, (const char*) block_content_copy);
					break;
				}
			}
		if (*flag == 1) {
			break;
		}
	}
}

int main(int argc, char **argv) {
	BYTE hashed_tx1[SHA256_HASH_SIZE], hashed_tx2[SHA256_HASH_SIZE], hashed_tx3[SHA256_HASH_SIZE], hashed_tx4[SHA256_HASH_SIZE],
			tx12[SHA256_HASH_SIZE * 2], tx34[SHA256_HASH_SIZE * 2], hashed_tx12[SHA256_HASH_SIZE], hashed_tx34[SHA256_HASH_SIZE],
			tx1234[SHA256_HASH_SIZE * 2], top_hash[SHA256_HASH_SIZE], block_content[BLOCK_SIZE];
	BYTE block_hash[SHA256_HASH_SIZE] = "0000000000000000000000000000000000000000000000000000000000000000"; // TODO: Update
	uint64_t nonce = 0; // TODO: Update
	size_t current_length;

	// Top hash
	apply_sha256(tx1, strlen((const char*)tx1), hashed_tx1, 1);
	apply_sha256(tx2, strlen((const char*)tx2), hashed_tx2, 1);
	apply_sha256(tx3, strlen((const char*)tx3), hashed_tx3, 1);
	apply_sha256(tx4, strlen((const char*)tx4), hashed_tx4, 1);
	strcpy((char *)tx12, (const char *)hashed_tx1);
	strcat((char *)tx12, (const char *)hashed_tx2);
	apply_sha256(tx12, strlen((const char*)tx12), hashed_tx12, 1);
	strcpy((char *)tx34, (const char *)hashed_tx3);
	strcat((char *)tx34, (const char *)hashed_tx4);
	apply_sha256(tx34, strlen((const char*)tx34), hashed_tx34, 1);
	strcpy((char *)tx1234, (const char *)hashed_tx12);
	strcat((char *)tx1234, (const char *)hashed_tx34);
	apply_sha256(tx1234, strlen((const char*)tx34), top_hash, 1);

	// prev_block_hash + top_hash
	strcpy((char*)block_content, (const char*)prev_block_hash);
	strcat((char*)block_content, (const char*)top_hash);
	current_length = strlen((char*) block_content);

	hipEvent_t start, stop;
	startTiming(&start, &stop);

	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, 0);
	const size_t block_size = 256;
	size_t blocks_no = props.multiProcessorCount;

	//alocate memory for device arrays and copy the information
	int number_th = blocks_no * block_size;
	BYTE *block_hash_d, *block_content_d, *difficulty_5_zeros_d;
	uint64_t *nonce_d;
	hipMalloc(&block_hash_d, SHA256_HASH_SIZE);
	if (block_hash_d == 0) {
		printf("[HOST]: Error allocating memory for block_hash_d\n");
		exit(1);
	}
	hipMalloc(&block_content_d, BLOCK_SIZE);
	if (block_content_d == 0) {
		printf("[HOST]: Error allocating memory for block_content_d\n");
		exit(1);
	}
	hipMemcpy(block_content_d, block_content, BLOCK_SIZE, hipMemcpyHostToDevice);
	hipMalloc(&nonce_d, sizeof(uint64_t));
	if (nonce_d == 0) {
		printf("[HOST]: Error allocating memory for nonce_d\n");
		exit(1);
	}
	hipMalloc(&difficulty_5_zeros_d, SHA256_HASH_SIZE);
	if (difficulty_5_zeros_d == 0) {
		printf("[HOST]: Error allocating memory for difficulty_5_zeros_d\n");
		exit(1);
	}
	hipMemcpy(difficulty_5_zeros_d, difficulty_5_zeros, SHA256_HASH_SIZE, hipMemcpyHostToDevice);
	int *flag;
	hipMalloc(&flag, sizeof(int));
	if (flag == 0) {
		printf("[HOST]: Error allocating memory for flag\n");
		exit(1);
	}
	hipMemset(flag, 0, sizeof(int));
	findNonce<<<blocks_no, block_size>>>(number_th, block_content_d, current_length, block_hash_d, nonce_d, difficulty_5_zeros_d, flag);
	hipDeviceSynchronize();
	//copy the results back to the host
	hipMemcpy(block_hash, block_hash_d, SHA256_HASH_SIZE, hipMemcpyDeviceToHost);
	hipMemcpy(block_content, block_content_d, BLOCK_SIZE, hipMemcpyDeviceToHost);
	hipMemcpy(&nonce, nonce_d, sizeof(uint64_t), hipMemcpyDeviceToHost);

	float seconds = stopTiming(&start, &stop);
	printResult(block_hash, nonce, seconds);

	// Free the memory for device
	hipFree(block_hash_d);
	hipFree(block_content_d);
	hipFree(nonce_d);
	hipFree(difficulty_5_zeros_d);
	hipFree(flag);
	return 0;
}
